#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
  printf("=== CUDA Device Query ===\n\n");

  int runtimeVersion = 0;
  CUDA_CHECK(hipRuntimeGetVersion(&runtimeVersion));
  printf("CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000,
         (runtimeVersion % 100) / 10);

  int driverVersion = 0;
  CUDA_CHECK(hipDriverGetVersion(&driverVersion));
  printf("CUDA Driver Version: %d.%d\n\n", driverVersion / 1000,
         (driverVersion % 100) / 10);

  int deviceCount = 0;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  if (deviceCount == 0) {
    fprintf(stderr, "Error: No CUDA-capable devices found!\n");
    return EXIT_FAILURE;
  }

  printf("Detected %d CUDA-capable device(s)\n\n", deviceCount);

  for (int dev = 0; dev < deviceCount; dev++) {
    printf("========================================\n");
    printf("Device %d: ", dev);

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev));

    printf("%s\n", prop.name);
    printf("========================================\n\n");

    printf("Compute Capability:\n");
    printf("  Major: %d\n", prop.major);
    printf("  Minor: %d\n", prop.minor);
    printf("  Version: %d.%d\n\n", prop.major, prop.minor);

    printf("Memory:\n");
    printf("  Total global memory: %.2f GB\n",
           prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    printf("  Total constant memory: %.2f KB\n", prop.totalConstMem / 1024.0);
    printf("  Shared memory per block: %.2f KB\n",
           prop.sharedMemPerBlock / 1024.0);
    printf("  L2 cache size: %.2f MB\n", prop.l2CacheSize / (1024.0 * 1024.0));
    printf("  Memory bus width: %d-bit\n\n", prop.memoryBusWidth);

    printf("Multiprocessors:\n");
    printf("  Number of SMs: %d\n", prop.multiProcessorCount);
    printf("  Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  Total CUDA cores: ~%d\n\n", prop.multiProcessorCount *
                                              prop.maxThreadsPerMultiProcessor /
                                              32 * 32);

    printf("Thread/Block Configuration:\n");
    printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Max block dimensions: [%d, %d, %d]\n", prop.maxThreadsDim[0],
           prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("  Max grid dimensions: [%d, %d, %d]\n", prop.maxGridSize[0],
           prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  Warp size: %d\n", prop.warpSize);
    printf("  Registers per block: %d\n", prop.regsPerBlock);
    printf("  Registers per SM: %d\n\n", prop.regsPerMultiprocessor);

    printf("Features:\n");
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "Yes" : "No");
    printf("  ECC enabled: %s\n", prop.ECCEnabled ? "Yes" : "No");
    printf("  Unified addressing: %s\n", prop.unifiedAddressing ? "Yes" : "No");
    printf("  Managed memory: %s\n", prop.managedMemory ? "Yes" : "No");
    printf("  Multi-GPU board: %s\n", prop.isMultiGpuBoard ? "Yes" : "No");
    printf("  Cooperative launch: %s\n", prop.cooperativeLaunch ? "Yes" : "No");
    printf("  Async engine count: %d\n", prop.asyncEngineCount);
    printf("  PCI bus ID: %d\n", prop.pciBusID);
    printf("  PCI device ID: %d\n\n", prop.pciDeviceID);
  }

  printf("========================================\n");
  printf("Device query completed successfully.\n");
  printf("========================================\n");

  return EXIT_SUCCESS;
}
